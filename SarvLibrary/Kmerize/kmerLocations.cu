#include "hip/hip_runtime.h"
/*  Written by Christopher Wright 11-3-2017
*   Program will take in input file in fasta format and use the gpu to count the number of kmers
*   Limitations: for a Nvidia k620 it works correctly up to kmer length of 14
*   Consistency: because of sparsity, the gpu kernel doesn't atomic add to make it faster, 
*                so there may be slight variations in the number of kmers counted
*   example of how to run:
*   nvcc -g -std=c++11 kmer_less14.cu 
*   time ./a.out 14 ~/Development/fasta_files/ecoli_mda_lan1_left.fasta
*
*   or if using clang:
*   export CUDA_FLAGS='-x cuda --cuda-gpu-arch=sm_50 -L/usr/local/cuda-9.0/lib64/ -lcudart_static -ldl -lrt -pthread'
*   clang++ -g -std=c++11 kmer_less14.cu $CUDA_FLAGS
*   time ./a.out 14 ~/Development/fasta_files/ecoli_mda_lan1_left.fasta
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <fcntl.h>
#include <string>
#include <map>
#include <unordered_map>
#include <vector>
#include <omp.h>
#include <cstddef>
#include <ctime>
#include "kmerLocations.h"

#define NUM_THREADS 128
#define NUM_BLOCKS 1024
#define MAX_seqLen 1024

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

struct GpuTimer{
        hipEvent_t start;
        hipEvent_t stop;
        GpuTimer(){
                hipEventCreate(&start);
                hipEventCreate(&stop);
        }
        ~GpuTimer(){
                hipEventDestroy(start);
                hipEventDestroy(stop);
        }
        void Start(){
                hipEventRecord(start, 0);
        }
        void Stop(){
                hipEventRecord(stop, 0);
        }
        float Elapsed(){
                float elapsed;
                hipEventSynchronize(stop);
                hipEventElapsedTime(&elapsed, start, stop);
                return elapsed;
        }
};

long line_count_kmerlocations(char const *fname){
    static const size_t BUFFER_SIZE = 16*1024;
    int fd = open(fname, O_RDONLY);
    if(fd == -1){
        printf("Error, open failed\n");
        exit(1);  
    }
    /* Advise the kernel of our access pattern.  */
    posix_fadvise(fd, 0, 0, 1);  // FDADVICE_SEQUENTIAL
    char buf[BUFFER_SIZE + 1];
    long lines = 0;
    while(size_t bytes_read = read(fd, buf, BUFFER_SIZE)){
        if(bytes_read == (size_t)-1){
            printf("Error, read failed\n");
        }
        if (!bytes_read)
            break;
        for(char *p = buf; (p = (char*) memchr(p, '\n', (buf + bytes_read) - p)); ++p)
            ++lines;
    }
    return lines;
}

int seqLength_kmerLocations(char const *fname){
    std::ifstream fd(fname);
    std::string line;
    std::getline(fd, line);
    if(line[0]=='>'){
        getline(fd, line);
    }
    return line.length();
}

inline std::string getFileToString_kmerLocations(const std::string filename){
    std::string toReturn = "";
    std::string line;
    std::ifstream fp(filename.c_str());
    std::getline(fp,line);//don't care about the first line
    while(std::getline(fp,line)){
        toReturn+=line;
        std::getline(fp,line);
    }
    return toReturn;
}


__global__ void D_GPUKmerizeLocations(   const unsigned char* d_input, uint64_t* d_keys, int* d_locations, const int seqLen, 
                                const int kmerLen, const int numSeqPerBlock, const int seqLimit, const int startingIndex){
    const uint8_t char_values[] = {0,0,1,0,0,0,2,0,0,0,0,0,0,0,0,0,0,0,0,3};
    __shared__ unsigned char sd_input[NUM_THREADS];//only need seqLen, but need a static size
    unsigned int g_start;
    uint64_t hash;
    uint8_t c;
    for(int i = 0; i < numSeqPerBlock; ++i){
        int seqId = blockIdx.x * numSeqPerBlock + i;
        if (seqId < seqLimit) {
            g_start = seqId * seqLen;
            if (threadIdx.x < seqLen) {
                sd_input[threadIdx.x] = d_input[g_start + threadIdx.x];
            }
            __syncthreads();
            if (threadIdx.x <= (seqLen - kmerLen)) {
                hash = 0;
                for(int k = threadIdx.x; k < threadIdx.x + kmerLen; ++k){
                    c = sd_input[k];
                    hash = hash << 2;
                    hash |= char_values[(c-65)];
                }
                d_keys[seqId* (seqLen-kmerLen + 1) + threadIdx.x] = hash;
                d_locations[seqId* (seqLen-kmerLen + 1) + threadIdx.x] = startingIndex + i;
            }
            __syncthreads();
        }
    }
}

void H_GPUKmerizeLocations(const std::string inFilename, const int kmerLen, std::unordered_map<uint64_t, std::vector<unsigned int> >& kmerMap) {
    const int numSeq = line_count_kmerlocations(inFilename.c_str())/2;
    const std::string inSeqString = getFileToString_kmerLocations(inFilename);
    const char *h_input = inSeqString.c_str();
    const int seqLen = seqLength_kmerLocations(inFilename.c_str());
    if(kmerLen > seqLen){
        printf("Desired Kmer Length is longer than the sequence length, exiting\n");
        exit(1);
    }
    unsigned char* d_input;//need to send data as well: d_input
    uint64_t* d_keys;
    int *d_locations;
    size_t FREE, TOTAL;

    checkCudaErrors(hipMemGetInfo(&FREE,&TOTAL));
    size_t USABLE_GPU_MEM = FREE * 0.95;//Only use 95% of total memory
    size_t entriesPerSeq = seqLen - kmerLen + 1;
    size_t bytesPerEntry = sizeof(uint64_t) + sizeof(int);//for the sizeof(key) + sizeof(location)
    size_t bytesInputSeq = seqLen;
    size_t bytesPerSeq = (entriesPerSeq * bytesPerEntry) + bytesInputSeq; 
    size_t numSeqPerBlock = USABLE_GPU_MEM / (bytesPerSeq * NUM_BLOCKS);
    size_t numSeqPerIter = numSeqPerBlock * NUM_BLOCKS;
    size_t arraySize = entriesPerSeq * numSeqPerIter;
    size_t gpuInputSize = (seqLen*numSeqPerIter) > inSeqString.size() ? inSeqString.size() : (seqLen*numSeqPerIter);
    size_t numIters = numSeq / numSeqPerIter + 1;

    printf("numSeq: %d\n", numSeq);
    printf("seqLen: %d\n", seqLen);
    printf("kmerLen: %d\n", kmerLen);
    printf("FREE: %lu\n", FREE);
    printf("TOTAL: %lu\n", TOTAL);
    printf("USABLE_GPU_MEM: %lu\n", USABLE_GPU_MEM);
    printf("entriesPerSeq: %lu\n", entriesPerSeq);
    printf("bytesPerEntry: %lu\n", bytesPerEntry);
    printf("bytesInputSeq: %lu\n", bytesInputSeq);
    printf("bytesPerSeq: %lu\n", bytesPerSeq);
    printf("numSeqPerBlock: %lu\n", numSeqPerBlock);
    printf("numSeqPerIter: %lu\n", numSeqPerIter);
    printf("arraySize: %lu\n", arraySize);
    printf("gpuInputSize: %lu\n", gpuInputSize);
    printf("numIters: %lu\n", numIters);
    printf("Using %lu / %lu bytes on GPU\n", numSeqPerIter * bytesPerSeq  + TOTAL - FREE, TOTAL);
    uint64_t* h_keys = new uint64_t[arraySize];
    int* h_locations = new int[arraySize];
    
    GpuTimer timer;timer.Start();
    double insertTotal = 0.0;
    checkCudaErrors(hipMalloc(&d_input, gpuInputSize));
    checkCudaErrors(hipMalloc(&d_keys, arraySize * sizeof(uint64_t)));
    checkCudaErrors(hipMalloc(&d_locations, arraySize * sizeof(int)));

    int currSize = 0;
    int numSeqProcessed = 0;
    clock_t start, end;
    //TODO:
    //make array of streams
    //make array of multimaps to use
    //implement each loop as working on a stream

    for(int i = 0; i < numIters; ++i){
        if ((i+1) * numSeqPerIter < numSeq)
            currSize = numSeqPerIter;
        else 
            currSize = numSeq - (i * numSeqPerIter);
        
        printf("Processed %d/%d sequences\n", numSeqProcessed, numSeq);
        numSeqProcessed += currSize;
        int startingIndex = i * gpuInputSize;
        checkCudaErrors(hipMemcpy(d_input, &(h_input[startingIndex]), currSize * seqLen, hipMemcpyHostToDevice));
        // checkCudaErrors(hipMemset(d_keys, 0, seqLen * currSize)); //don't need to memset, we overwrite in the kernel
        D_GPUKmerizeLocations<<<NUM_BLOCKS, NUM_THREADS>>>(d_input, d_keys, d_locations, seqLen, kmerLen, numSeqPerBlock, currSize, startingIndex);
        hipDeviceSynchronize();
        checkCudaErrors(hipMemcpy(h_keys, d_keys, entriesPerSeq * currSize * sizeof(uint64_t), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_locations, d_locations, entriesPerSeq * currSize * sizeof(int), hipMemcpyDeviceToHost));

        // printf("Inserting into kmerMap\n");
        start = clock();
        for(int i = 0; i < entriesPerSeq * currSize; ++i){
            // counter++;
            kmerMap[h_keys[i]].push_back(h_locations[i]);
            // kmerMap.insert(std::pair<uint64_t, int>(h_keys[i], h_locations[i]));
        }
        end = clock();
        // printf("Time spent on insertion into map this loop: %f\n", ((double) (end - start)) / CLOCKS_PER_SEC);
        insertTotal += ((double) (end - start)) / CLOCKS_PER_SEC;
    }
    // printf("Total time spent just in insertion: %f\n", insertTotal);
    // printf("Total number of kmers: %zu\n", kmerMap.size());
    printf("Processed %d/%d sequences\n", numSeqProcessed, numSeq);
    // printf("Getting final number of unique kmers\n");
    // long num_kmers = h_countsMap.size();
    // std::ofstream out(outFilename.c_str());
    // for(int i = 0; i < (int)(pow(4.0, double(kmerLen))); ++i){
    //     if(h_counts[i] > 0){
    //         num_kmers++;
    //         // out<<">"<<int(h_counts[i])<<"\n";
    //         out<<i<<"\n";
    //     }
    // }
    // out.close();
    // printf("Number of kmers gpu is %ld\n", num_kmers);
    checkCudaErrors(hipFree(d_keys));
    checkCudaErrors(hipFree(d_locations));
    checkCudaErrors(hipFree(d_input));
    timer.Stop();
    printf("Finished Kmer Location GPU code in %f seconds\n", timer.Elapsed()/1000);
}
namespace sarv{
    void GPUKmerizeLocationWrapper(std::string inFilename, unsigned int kmerLen, std::unordered_map<uint64_t, std::vector<unsigned int> >& kmerMap){
        H_GPUKmerizeLocations(inFilename, kmerLen, kmerMap);
    }
}

// int main(int argc, char* argv[]){
//     printf("Usage: ./kmerLocs14 <kmerLen> <inputFilename>\n");
//     const int kmerLen = atoi(argv[1]);
//     std::string inFilename(argv[2]);
//     std::unordered_map<uint64_t, std::vector<unsigned int> > kmerMap;
//     GPUKmerizeLocationWrapper(inFilename, kmerLen, kmerMap);
//     return 0;
// }