#include "hip/hip_runtime.h"
/*  Written by Christopher Wright 11-3-2017
*   Program will take in input file in fasta format and use the gpu to count the number of kmers
*   Limitations: for a Nvidia k620 it works correctly up to kmer length of 14
*   Consistency: because of sparsity, the gpu kernel doesn't atomic add to make it faster, 
*                so there may be slight variations in the number of kmers counted
*   example of how to run:
*   nvcc -g -std=c++11 kmer_less14.cu 
*   time ./a.out 14 ~/Development/fasta_files/ecoli_mda_lan1_left.fasta
*
*   or if using clang:
*   export CUDA_FLAGS='-x cuda --cuda-gpu-arch=sm_50 -L/usr/local/cuda-9.0/lib64/ -lcudart_static -ldl -lrt -pthread'
*   clang++ -g -std=c++11 kmer_less14.cu $CUDA_FLAGS
*   time ./a.out 14 ~/Development/fasta_files/ecoli_mda_lan1_left.fasta
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <fcntl.h>
#include <string>
#include <unordered_map>
#include <set>
#include <unordered_set>
#include <array>
#include <sstream>
#define NUM_THREADS 128
#define NUM_BLOCKS 1024
#define MAX_seqLen 1024
#include "cuda128t256t.h"
#include <omp.h>
#include <cstddef>
#include <sys/wait.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <memory>
#include <stdexcept>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

struct GpuTimer{
        hipEvent_t start;
        hipEvent_t stop;
        GpuTimer(){
                hipEventCreate(&start);
                hipEventCreate(&stop);
        }
        ~GpuTimer(){
                hipEventDestroy(start);
                hipEventDestroy(stop);
        }
        void Start(){
                hipEventRecord(start, 0);
        }
        void Stop(){
                hipEventRecord(stop, 0);
        }
        float Elapsed(){
                float elapsed;
                hipEventSynchronize(stop);
                hipEventElapsedTime(&elapsed, start, stop);
                return elapsed;
        }
};

std::string exec_com(const char* cmd) {
    std::array<char, 128> buffer;
    std::string result;
    std::unique_ptr<FILE, decltype(&pclose)> pipe(popen(cmd, "r"), pclose);
    while (fgets(buffer.data(), buffer.size(), pipe.get()) != nullptr) {
        result += buffer.data();
    }
    return result;
}

long line_count(char const *fname){
    std::string cmd = "wc -l " + std::string(fname);
    std::string result = exec_com(cmd.c_str());
    std::stringstream ss(result);
    ss>>result;
    int nlines = std::stoi(result);
    return nlines;
}

int seqLength(char const *fname){
    std::ifstream fd(fname);
    std::string line;
    std::getline(fd, line);
    if(line[0]=='>'){
        getline(fd, line);
        return line.length();
    }
    printf("SeqLen returned is 0, error!\n");
    return 0;
}

inline std::string getFileToString(const std::string filename){
    std::string toReturn = "";
    std::string line;
    std::ifstream fp(filename.c_str());
    std::getline(fp,line);//don't care about the first line
    while(std::getline(fp,line)){
        toReturn+=line;
        std::getline(fp,line);
    }
    return toReturn;
}

inline void PrintIntToKmer(int key, const int kmerLen, char* KArray){
    for(int i = 0; i < kmerLen; ++i){
        switch(key & 0x00000003){
            case 0:
                KArray[kmerLen - 1 - i] = 'A';
                break;
            case 1:
                KArray[kmerLen - 1 - i] = 'C';
                break;
            case 2:
                KArray[kmerLen - 1 - i] = 'G';
                break;
            case 3:
                KArray[kmerLen - 1 - i] = 'T';
                break;
            default:
                KArray[kmerLen - 1 - i] = 'A';
                break;
        }
        key = key >> 2;
    }
}

__global__ void D_GPUKmerize(   const unsigned char* d_input, uint8_t* d_counts, const int seqLen, 
								const int kmerLen, const int numSeqPerBlock, const int seqLimit){
    const uint8_t char_values[] = {0,0,1,0,0,0,2,0,0,0,0,0,0,0,0,0,0,0,0,3};
    __shared__ unsigned char sd_input[NUM_THREADS];//only need seqLen, but need a static size
    unsigned int g_start;
    int hash;
    uint8_t c;
    for(int i = 0; i < numSeqPerBlock; ++i){
        int seqId = blockIdx.x * numSeqPerBlock + i;
        if (seqId < seqLimit) {
            g_start = (blockIdx.x * numSeqPerBlock + i) * seqLen;
            if (threadIdx.x < seqLen) {
                sd_input[threadIdx.x] = d_input[g_start + threadIdx.x];
            }
            __syncthreads();
            if (threadIdx.x <= (seqLen - kmerLen)) {
                hash = 0;
                for(int k = threadIdx.x; k < threadIdx.x + kmerLen; ++k){
                    c = sd_input[k];
                    hash = hash << 2;
                    hash |= char_values[(c-65)];
                }
                if(d_counts[hash] < 255)
                    d_counts[hash]++;
            }
            __syncthreads();
        }
    }
}
#define GPUOUTSIZE 268435456
// #define BYTES_USED_GPULOCAL (NUM_THREADS * NUM_BLOCKS * 100 + 268435456)
void H_GPUKmerize(const std::string inFilename, const int kmerLen, std::string outFilename){
    size_t arraySize = (size_t)(pow(4.0, double(kmerLen)));   
    uint8_t* kmerCounts = new uint8_t[arraySize];
    std::fill(kmerCounts, kmerCounts + arraySize, 0);
    const int numSeq = line_count(inFilename.c_str())/2;
    std::string inSeqString = getFileToString(inFilename);
    const char *h_input = inSeqString.c_str();
    int seqLen = seqLength(inFilename.c_str());
    unsigned char* d_input;
    uint8_t* d_counts;
    size_t FREE, TOTAL;
    checkCudaErrors(hipMemGetInfo(&FREE,&TOTAL));
    size_t USABLE_GPU_MEM = (FREE - GPUOUTSIZE) * 0.90;
    // FREE -= BYTES_USED_GPULOCAL; 
    size_t numSeqPerIter = USABLE_GPU_MEM / seqLen;
    size_t numSeqPerBlock = numSeqPerIter / NUM_BLOCKS;
    size_t numIters = numSeq / numSeqPerIter + 1;
    size_t gpuInputSize = (seqLen*numSeqPerIter) > inSeqString.size() ? inSeqString.size() : (seqLen*numSeqPerIter);
    uint8_t* h_counts = new uint8_t[arraySize];
    printf("numSeq: %d\n", numSeq);
    printf("seqLen: %d\n", seqLen);
    printf("FREE: %lu\n", FREE);
    printf("TOTAL: %lu\n", TOTAL);
    printf("numSeqPerBlock: %lu\n", numSeqPerBlock);
    printf("numSeqPerIter: %lu\n", numSeqPerIter);
    printf("numIters: %lu\n", numIters);
    printf("arraySize: %lu\n", arraySize);
    printf("gpuInputSize: %lu\n", gpuInputSize);
    printf("Using %lu / %lu bytes on GPU\n", gpuInputSize + GPUOUTSIZE + TOTAL - FREE, TOTAL);
    fflush(stdout);
    
    /********CUDA MALLOC/MEMSET/MEMCPY********************/
    checkCudaErrors(hipMalloc(&d_input, gpuInputSize));
    checkCudaErrors(hipMalloc(&d_counts, arraySize*sizeof(uint8_t)));
    checkCudaErrors(hipMemset(d_counts, 0, arraySize*sizeof(uint8_t)));
    /********END OF CUDA MALLOC/MEMSET/MEMCPY*************/
    int currSize = 0;
    int numSeqProcessed = 0;
    for(int i = 0; i < numIters; ++i){
        if ((i+1) * numSeqPerIter < numSeq)
            currSize = numSeqPerIter;
        else 
            currSize = numSeq - (i * numSeqPerIter);

        printf("Processed %d/%d sequences\n", numSeqProcessed, numSeq);
        numSeqProcessed += currSize;
        checkCudaErrors(hipMemcpy(d_input, &(h_input[i * gpuInputSize]), seqLen * currSize, hipMemcpyHostToDevice));
        D_GPUKmerize<<<NUM_BLOCKS, NUM_THREADS>>>(d_input, d_counts, seqLen, kmerLen, numSeqPerBlock, currSize);
        hipDeviceSynchronize();
    }
    printf("Processed %d/%d sequences\n", numSeq, numSeq);
    checkCudaErrors(hipMemcpy(h_counts, d_counts, arraySize*sizeof(uint8_t), hipMemcpyDeviceToHost));
    long num_kmers = 0;
    printf("Getting final number of unique kmers\n");
    std::stringstream ss;
    std::ofstream out(outFilename.c_str());
    #pragma omp parallel for reduction(+:num_kmers)
    for(int i = 0; i < arraySize; ++i){
        if(h_counts[i] > 0){
            num_kmers++;
            std::string toPrint = ">";
            toPrint += std::to_string(int(h_counts[i])) + '\n' + std::to_string(i) + '\n';
            #pragma omp critical
            out<<toPrint;
            // ss<<">"<<int(h_counts[i])<<"\n";
            // ss<<i<<"\n";
        }
    }
    
    // out<<ss.rdbuf();
    out.close();
    printf("Number of unique kmers gpu is %ld\n", num_kmers);
    checkCudaErrors(hipFree(d_counts));
    checkCudaErrors(hipFree(d_input));
}

int main(int argc, char** argv){
    // printf("Usage: ./a.out <kmerLength> <inputfilename> <outputfilename>\n");
    const int kmerLen = atoi(argv[1]);
    std::string inFilename(argv[2]);
    std::string outFilename(argv[3]);
    
    GpuTimer timer;
    timer.Start();
    H_GPUKmerize(inFilename, kmerLen, outFilename);
    timer.Stop();
    printf("Finished Kmer Count GPU code in %f seconds\n", timer.Elapsed()/1000);

    return 0;
}
