#include "hip/hip_runtime.h"
/*  Written by Christopher Wright 11-3-2017
*   Program will take in input file in fasta format and use the gpu to count the number of kmers
*   Limitations: for a Nvidia k620 it works correctly up to kmer length of 14
*   Consistency: because of sparsity, the gpu kernel doesn't atomic add to make it faster, 
*                so there may be slight variations in the number of kmers counted
*   example of how to run:
*   nvcc -g -std=c++11 kmer_less14.cu 
*   time ./a.out 14 ~/Development/fasta_files/ecoli_mda_lan1_left.fasta
*
*   or if using clang:
*   export CUDA_FLAGS='-x cuda --cuda-gpu-arch=sm_50 -L/usr/local/cuda-9.0/lib64/ -lcudart_static -ldl -lrt -pthread'
*   clang++ -g -std=c++11 kmer_less14.cu $CUDA_FLAGS
*   time ./a.out 14 ~/Development/fasta_files/ecoli_mda_lan1_left.fasta
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <fcntl.h>
#include <string>
#include <unordered_map>
#include <omp.h>
#include <cstddef>
// #include "tbb/concurrent_unordered_map.h"
// #include "tbb/concurrent_hash_map.h"
// #include "tbb/blocked_range.h"
// #include "tbb/parallel_for.h"

#include "cuda128t256t.h"
#define NUM_THREADS 128
#define NUM_BLOCKS 1024
#define MAX_seqLen 1024

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

struct GpuTimer{
        hipEvent_t start;
        hipEvent_t stop;
        GpuTimer(){
                hipEventCreate(&start);
                hipEventCreate(&stop);
        }
        ~GpuTimer(){
                hipEventDestroy(start);
                hipEventDestroy(stop);
        }
        void Start(){
                hipEventRecord(start, 0);
        }
        void Stop(){
                hipEventRecord(stop, 0);
        }
        float Elapsed(){
                float elapsed;
                hipEventSynchronize(stop);
                hipEventElapsedTime(&elapsed, start, stop);
                return elapsed;
        }
};

long line_count(char const *fname){
    static const size_t BUFFER_SIZE = 16*1024;
    int fd = open(fname, O_RDONLY);
    if(fd == -1){
        printf("Error, open failed\n");
        exit(1);  
    }
    /* Advise the kernel of our access pattern.  */
    posix_fadvise(fd, 0, 0, 1);  // FDADVICE_SEQUENTIAL
    char buf[BUFFER_SIZE + 1];
    long lines = 0;
    while(size_t bytes_read = read(fd, buf, BUFFER_SIZE)){
        if(bytes_read == (size_t)-1){
            printf("Error, read failed\n");
        }
        if (!bytes_read)
            break;
        for(char *p = buf; (p = (char*) memchr(p, '\n', (buf + bytes_read) - p)); ++p)
            ++lines;
    }
    return lines;
}

int seqLength(char const *fname){
    std::ifstream fd(fname);
    std::string line;
    std::getline(fd, line);
    if(line[0]=='>'){
        getline(fd, line);
    }
    return line.length();
}

inline std::string getFileToString(const std::string filename){
    std::string toReturn = "";
    std::string line;
    std::ifstream fp(filename.c_str());
    std::getline(fp,line);//don't care about the first line
    while(std::getline(fp,line)){
        toReturn+=line;
        std::getline(fp,line);
    }
    return toReturn;
}


__global__ void D_GPUKmerize256(   const unsigned char* d_input, uint256_t* d_keys, const int seqLen, 
                                const int kmerLen, const int numSeqPerBlock, const int seqLimit){
    const uint8_t char_values[] = {0,0,1,0,0,0,2,0,0,0,0,0,0,0,0,0,0,0,0,3};
    __shared__ unsigned char sd_input[NUM_THREADS];//only need seqLen, but need a static size
    unsigned int g_start;
    uint256_t hash;
    uint8_t c;
    for(int i = 0; i < numSeqPerBlock; ++i){
        int seqId = blockIdx.x * numSeqPerBlock + i;
        if (seqId < seqLimit) {
            g_start = seqId * seqLen;
            if (threadIdx.x < seqLen) {
                sd_input[threadIdx.x] = d_input[g_start + threadIdx.x];
            }
            __syncthreads();
            if (threadIdx.x <= (seqLen - kmerLen)) {
                hash = 0;
                for(int k = threadIdx.x; k < threadIdx.x + kmerLen; ++k){
                    c = sd_input[k];
                    hash = hash << 2;
                    hash |= char_values[(c-65)];
                }
                d_keys[seqId* (seqLen-kmerLen + 1) + threadIdx.x] = hash;
            }
            __syncthreads();
        }
    }
}

//Good for kmerLen <=14 on my machine
__global__ void D_GPUKmerize(   const unsigned char* d_input, unsigned int* d_keys, const int seqLen, 
                                const int kmerLen, const int numSeqPerBlock, const int seqLimit){
    const uint8_t char_values[] = {0,0,1,0,0,0,2,0,0,0,0,0,0,0,0,0,0,0,0,3};
    __shared__ unsigned char sd_input[NUM_THREADS];//only need seqLen, but need a static size
    unsigned int g_start;
    unsigned int hash;
    uint8_t c;
    for(int i = 0; i < numSeqPerBlock; ++i){
        int seqId = blockIdx.x * numSeqPerBlock + i;
        if (seqId < seqLimit) {
            g_start = seqId * seqLen;
            if (threadIdx.x < seqLen) {
                sd_input[threadIdx.x] = d_input[g_start + threadIdx.x];
            }
            __syncthreads();
            if (threadIdx.x <= (seqLen - kmerLen)) {
                hash = 0;
                for(int k = threadIdx.x; k < threadIdx.x + kmerLen; ++k){
                    c = sd_input[k];
                    hash = hash << 2;
                    hash |= char_values[(c-65)];
                }
                d_keys[seqId* (seqLen-kmerLen + 1) + threadIdx.x] = hash;
            }
            __syncthreads();
        }
    }
}

void H_GPUKmerize(const std::string inFilename, const int kmerLen, std::string outFilename) {
    // typedef tbb::concurrent_unordered_map<uint256_t, int, uint256_tHasher> hash_t;
    // typedef tbb::concurrent_unordered_map<uint256_t, int, uint256_tHasher> hash_t;
    typedef std::unordered_map<uint256_t, int, uint256_tHasher> hash_t;
    // typedef std::unordered_map<uint64_t, int> hash_t;

    // uint8_t* h_counts = new uint8_t[(int)(pow(4.0, double(kmerLen)))];
    // std::fill(h_counts, h_counts + (int)(pow(4.0, double(kmerLen))), 0); 
    hash_t h_countsMap; 
    const int numSeq = line_count(inFilename.c_str())/2;
    const std::string inSeqString = getFileToString(inFilename);
    const char *h_input = inSeqString.c_str();
    const int seqLen = seqLength(inFilename.c_str());
    if(kmerLen > seqLen){
        printf("Desired Kmer Length is longer than the sequence length, exiting\n");
        exit(1);
    }
    unsigned char* d_input;//need to send data as well: d_input
    uint256_t* d_keys;
    size_t FREE, TOTAL;

    checkCudaErrors(hipMemGetInfo(&FREE,&TOTAL));
    size_t USABLE_GPU_MEM = FREE * 0.95;//Only use 95% of total memory
    size_t keysPerSeq = seqLen - kmerLen + 1;
    size_t bytesPerKey = sizeof(uint256_t);
    size_t bytesInputSeq = seqLen;
    size_t bytesPerSeq = keysPerSeq * bytesPerKey + bytesInputSeq;
    size_t numSeqPerBlock = USABLE_GPU_MEM / (bytesPerSeq * NUM_BLOCKS);
    size_t numSeqPerIter = numSeqPerBlock * NUM_BLOCKS;
    size_t arraySize = keysPerSeq * numSeqPerIter;
    size_t gpuInputSize = (seqLen*numSeqPerIter) > inSeqString.size() ? inSeqString.size() : (seqLen*numSeqPerIter);
    size_t numIters = numSeq / numSeqPerIter + 1;

    printf("numSeq: %d\n", numSeq);
    printf("seqLen: %d\n", seqLen);
    printf("kmerLen: %d\n", kmerLen);
    printf("FREE: %lu\n", FREE);
    printf("TOTAL: %lu\n", TOTAL);
    printf("USABLE_GPU_MEM: %lu\n", USABLE_GPU_MEM);
    printf("keysPerSeq: %lu\n", keysPerSeq);
    printf("bytesPerKey: %lu\n", bytesPerKey);
    printf("bytesInputSeq: %lu\n", bytesInputSeq);
    printf("bytesPerSeq: %lu\n", bytesPerSeq);
    printf("numSeqPerBlock: %lu\n", numSeqPerBlock);
    printf("numSeqPerIter: %lu\n", numSeqPerIter);
    printf("arraySize: %lu\n", arraySize);
    printf("gpuInputSize: %lu\n", gpuInputSize);
    printf("numIters: %lu\n", numIters);
    printf("Using %lu / %lu bytes on GPU\n", numSeqPerIter * bytesPerSeq  + TOTAL - FREE, TOTAL);
    uint256_t* h_keys = new uint256_t[arraySize];
    
    GpuTimer timer;timer.Start();
    checkCudaErrors(hipMalloc(&d_input, gpuInputSize));
    checkCudaErrors(hipMalloc(&d_keys, arraySize * sizeof(uint256_t)));

    long counter = 0;
    int currSize = 0;
    int numSeqProcessed = 0;
    for(int i = 0; i < numIters; ++i){
        if ((i+1) * numSeqPerIter < numSeq)
            currSize = numSeqPerIter;
        else 
            currSize = numSeq - (i * numSeqPerIter);
        
        printf("Processed %d/%d sequences\n", numSeqProcessed, numSeq);
        numSeqProcessed += currSize;
        checkCudaErrors(hipMemcpy(d_input, &(h_input[i * gpuInputSize]), currSize * seqLen, hipMemcpyHostToDevice));
        // checkCudaErrors(hipMemset(d_keys, 0, seqLen * currSize)); //don't need to memset, we overwrite in the kernel
        D_GPUKmerize256<<<NUM_BLOCKS, NUM_THREADS>>>(d_input, d_keys, seqLen, kmerLen, numSeqPerBlock, currSize);
        // D_GPUKmerize<<<NUM_BLOCKS, NUM_THREADS>>>(d_input, d_keys, seqLen, kmerLen, numSeqPerBlock, currSize);
        hipDeviceSynchronize();
        checkCudaErrors(hipMemcpy(h_keys, d_keys, keysPerSeq * currSize * sizeof(uint256_t), hipMemcpyDeviceToHost));

        #pragma omp parallel for reduction(+:counter)
        for(int i = 0; i < keysPerSeq * currSize; ++i){
            counter++;
            // h_countsMap[h_keys[i]]++;
            // auto exists = h_countsMap.find(h_keys[i]);
            // if(exists != h_countsMap.end())
            //     exists->second++;
            // else
            //     h_countsMap.insert(std::make_pair(h_keys[i], 1));
        }
    }
    printf("Total number of kmers: %lu\n", counter);
    printf("Processed %d/%d sequences\n", numSeqProcessed, numSeq);
    printf("Getting final number of unique kmers\n");
    long num_kmers = h_countsMap.size();
    // std::ofstream out(outFilename.c_str());
    // for(int i = 0; i < (int)(pow(4.0, double(kmerLen))); ++i){
    //     if(h_counts[i] > 0){
    //         num_kmers++;
    //         // out<<">"<<int(h_counts[i])<<"\n";
    //         out<<i<<"\n";
    //     }
    // }
    // out.close();
    printf("Number of kmers gpu is %ld\n", num_kmers);
    checkCudaErrors(hipFree(d_keys));
    checkCudaErrors(hipFree(d_input));
    timer.Stop();
    printf("Finished Kmer Count GPU code in %f seconds\n", timer.Elapsed()/1000);
}

int main(int argc, char** argv){
    // printf("Usage: ./a.out <kmerLength> <inputfilename> <outputfilename>\n");
    const int kmerLen = atoi(argv[1]);
    std::string inFilename(argv[2]);
    std::string outFilename(argv[3]);
    H_GPUKmerize(inFilename, kmerLen, outFilename);

    return 0;
}
